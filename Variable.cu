#include "hip/hip_runtime.h"
#include "Variable.h"

Variable::Variable (const int Nx, const int Ny) {
    // Allocate on Host
    h_var  = (double*)malloc(Nx*Ny*sizeof(double));
    h_varx = (double*)malloc(Nx*Ny*sizeof(double));
    h_vary = (double*)malloc(Nx*Ny*sizeof(double));
    // Allocate on Device
    hipMalloc(&d_var,Nx*Ny*sizeof(double));
    hipMalloc(&d_varx,Nx*Ny*sizeof(double));
    hipMalloc(&d_vary,Nx*Ny*sizeof(double));
    hipMalloc(&VAR,(Nx/2+1)*Ny*sizeof(hipDoubleComplex));
    hipMalloc(&VARX,(Nx/2+1)*Ny*sizeof(hipDoubleComplex));
    hipMalloc(&VARY,(Nx/2+1)*Ny*sizeof(hipDoubleComplex));
    // CUFFT Plans
    hipfftPlan2d(&fft_var, Ny, Nx, HIPFFT_D2Z);
    hipfftPlan2d(&ifft_VARX, Ny, Nx, HIPFFT_Z2D);
    hipfftPlan2d(&ifft_VARY, Ny, Nx, HIPFFT_Z2D);
    hipfftPlan2d(&ifft_VAR, Ny, Nx, HIPFFT_Z2D);
};

Variable::~Variable () {
    hipfftDestroy(fft_var);
    hipfftDestroy(ifft_VARX);
    hipfftDestroy(ifft_VARY);
    free(h_var);
    free(h_varx);
    free(h_vary);
    hipFree(d_var);
    hipFree(d_varx);
    hipFree(d_vary);
    hipFree(VAR);
    hipFree(VARX);
    hipFree(VARY);
};

double Variable::MaxVar (const int Nx, const int Ny) {
    // Calculate the maximum velocity for adaptive dt
    double max_var = fabs(h_var[0]);
    for (int ii=1; ii<Nx*Ny; ii++) {
        if (max(fabs(h_var[ii]),max_var) != max_var) {
            max_var = fabs(h_var[ii]);
        }
    }
    return max_var;
}

void Variable::ComputeDerivatives (const double* k, const double* l, const int Nx, const int Ny, const int tpb, const int nblks) {
    // Fourier transform
    hipfftExecD2Z(fft_var, d_var, VAR);
    // Calculate derivatives
    CalcDerivs<<<nblks,tpb>>>(VAR, VARX, VARY, k, l, Nx, Ny);
    // Transform back
    hipfftExecZ2D(ifft_VARX, VARX, d_varx);
    hipfftExecZ2D(ifft_VARY, VARY, d_vary);
}

void Variable::Filter (const double* filter, const int Nx, const int Ny, const int tpb, const int nblks) {
    // Fourier transform
    hipfftExecD2Z(fft_var, d_var, VAR);
    // Apply filter
    takeFilter<<<nblks,tpb>>>(VAR, filter, Nx, Ny); 
    // Transform back     
    hipfftExecZ2D(ifft_VAR, VAR, d_var);
}