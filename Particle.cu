#include"Particle.h"

Particle::Particle(int in_num_particles) {
	// Pass number of particles to the Particle class
	num_particles = in_num_particles;
	// Allocate particle positions on the host
	h_part_pos_x = (double *)malloc(num_particles*sizeof(double));
    h_part_pos_y = (double *)malloc(num_particles*sizeof(double));
    // Allocate particle positions and interpolated velocities on device
    hipMalloc(&d_part_pos_x,num_particles*sizeof(double));
    hipMalloc(&d_part_pos_y,num_particles*sizeof(double));
    hipMalloc(&interp_u,num_particles*sizeof(double));
    hipMalloc(&interp_v,num_particles*sizeof(double));

}

Particle::~Particle() {
	// Free allocations
	hipFree(d_part_pos_x);
	hipFree(d_part_pos_y);
	hipFree(interp_u);
	hipFree(interp_v);
	free(h_part_pos_x);
	free(h_part_pos_y);
}